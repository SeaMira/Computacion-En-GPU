
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__ 
void matMul(const int N, float* A, float* B, float* C)
{
  __shared__ float Bwrk[1 << 10];

  int k;
	int j;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int iloc = threadIdx.x;
  int nloc = blockDim.x;
  float tmp = 0.0f;
  float Awrk[1 << 10];

  if (i >= N) return;
  
  for (k = 0; k < N; k++)
  {
    Awrk[k] = A[i * N + k];
  }

	for (j = 0; j < N; j++)
  {
    // Sincronizacion para almacenar la columna. Si son t threads por grupo que calcularian t filas, cada thread
    // se encarga de almacenar N / t valores de la columna actual en el arreglo local. Ej: Si son 3 threads por
    // grupo, el thread 0 de cada grupo se encarga de almacenar Bwrk[0], Bwrk[3], Bwrk[6], ...  
    for (k = iloc; k < N; k += nloc)
    {
      Bwrk[k] = B[k * N + j];
    }

    // Synchronize (ensure all the data is available)
    __syncthreads();

    tmp = 0.0f;
    for (k = 0; k < N; k++)
    {
      tmp += Awrk[k] * Bwrk[k];
    }
    C[i * N + j] = tmp;

    // Synchronize (ensure the j iteration was done by all threads before passing to next interation)
    __syncthreads();
  }
}

int main(void)
{
  // Matrices of 16K X 16K elements
  int N = 1 << 10;

  float* A = new float[N * N];
  float* B = new float[N * N];
  float* C = new float[N * N];

  // Initialize A and B matrices on the host
  for (int i = 0; i < N * N; i++)
  {
    A[i] = 1.0f;
    B[i] = 2.0f;
  }

	// Allocate device memory for matrices A, B, and C
	float *dA, *dB, *dC;
	hipMalloc((void**) &dA, N * N * sizeof(float));
	hipMalloc((void**) &dB, N * N * sizeof(float));
	hipMalloc((void**) &dC, N * N * sizeof(float));

	// Transfer matrices A and B from host to device
	hipMemcpy(dA, A, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dB, B, N * N * sizeof(float), hipMemcpyHostToDevice);

  // Blocks of size 256
  int blockSize = 256;

  // Round up in case N is not a multiple of blockSize
  int numBlocks = (N + blockSize - 1) / blockSize;

  // Run kernel on 1M elements on the GPU
	matMul<<<numBlocks, blockSize>>>(N, dA, dB, dC);

	// Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

	// Transfer matrix C from device to host
	hipMemcpy(C, dC, N * N * sizeof(float), hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

  // Check for errors (all values should be 32768.0f)
  float maxError = 0.0f;

  for (int i = 0; i < N * N; i++)
  {
    maxError = fmax(maxError, fabs(C[i] - 32768.0f));
  }
    
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  delete [] A;
  delete [] B;
  delete [] C;

  return 0;
}
