
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__ 
void matMul(const int N, float* A, float* B, float* C)
{
  int k;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  float tmp = 0.0f;

	if (i >= N || j >= N) return;
  
  for (k = 0; k < N; k++)
  {
    // Too many accesses to global memory
    tmp += A[i * N + k] * B[k * N + j];
  }

  C[i * N + j] = tmp;
}


int main(void)
{
  // Matrices of 16K X 16K elements
  int N = 1 << 14;

  float* A = new float[N * N];
  float* B = new float[N * N];
  float* C = new float[N * N];

  // Initialize A and B matrices on the host
  for (int i = 0; i < N * N; i++)
  {
    A[i] = 1.0f;
    B[i] = 2.0f;
  }

	// Allocate device memory for matrices A, B, and C
	float *dA, *dB, *dC;
	hipMalloc((void**) &dA, N * N * sizeof(float));
	hipMalloc((void**) &dB, N * N * sizeof(float));
	hipMalloc((void**) &dC, N * N * sizeof(float));

	// Transfer matrices A and B from host to device
	hipMemcpy(dA, A, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dB, B, N * N * sizeof(float), hipMemcpyHostToDevice);

  // Blocks of size 16 x 16
  int blockSize = 16;

  // Round up in case N is not a multiple of blockSize
  int numBlocks = (N + blockSize - 1) / blockSize;

	// Define block and grid dimensions
	dim3 blockDim(blockSize, blockSize);
	dim3 gridDim(numBlocks, numBlocks);

  // Run kernel on 1M elements on the GPU
	matMul<<<gridDim, blockDim>>>(N, dA, dB, dC);

	// Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

	// Transfer matrix C from device to host
	hipMemcpy(C, dC, N * N * sizeof(float), hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

  // Check for errors (all values should be 32768.0f)
  float maxError = 0.0f;

  for (int i = 0; i < N * N; i++)
  {
    maxError = fmax(maxError, fabs(C[i] - 32768.0f));
  }

  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  delete [] A;
  delete [] B;
  delete [] C;

  return 0;
}